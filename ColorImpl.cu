#include "hip/hip_runtime.h"
#include "ColorImpl.cuh"

static double lerp(double v0, double v1, double t) {
    return (1 - t) * v0 + t * v1;
}

static PPMPixel lerp_ppm(PPMPixel a, PPMPixel b, double t) {
    PPMPixel out;
    out.red = static_cast<unsigned char>(round(lerp(a.red, b.red, t)));
    out.green = static_cast<unsigned char>(round(lerp(a.green, b.green, t)));
    out.blue = static_cast<unsigned char>(round(lerp(a.blue, b.blue, t)));
    return out;
}

PPMPixel GetColor(double proportion) {
    if (proportion == 0) {
        PPMPixel result = {0, 0, 0};
        return result;
    }

    PPMPixel point0 = {255, 255, 255};
    PPMPixel point1 = {128, 0, 0};
    PPMPixel point2 = {120, 81, 169};
    PPMPixel point3 = {0, 0, 128};

    PPMPixel a0 = lerp_ppm(point0, point1, proportion);
    PPMPixel a1 = lerp_ppm(point1, point2, proportion);
    PPMPixel a2 = lerp_ppm(point2, point3, proportion);

    PPMPixel b0 = lerp_ppm(a0, a1, proportion);
    PPMPixel b1 = lerp_ppm(a1, a2, proportion);

    return lerp_ppm(b0, b1, proportion);
}
