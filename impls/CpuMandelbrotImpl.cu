#include "hip/hip_runtime.h"
#include "CpuMandelbrotImpl.cuh"
#include "../ColorImpl.cuh"

static constexpr size_t MAX_ITERATION = 32;
static constexpr double X_MIN = -2.01;
static constexpr double X_MAX = 0.48;
static constexpr double Y_MIN = -1.13;
static constexpr double Y_MAX = 1.13;

static unsigned int GetIterations(double x0, double y0) {
    double x = 0;
    double y = 0;
    int iteration = 0;
    while (x * x + y * y <= 4 && iteration < MAX_ITERATION) {
        double x_temp = x * x - y * y + x0;
        y = 2 * x * y + y0;
        x = x_temp;
        iteration++;
    }
    return iteration;
}

std::chrono::microseconds CpuMandelbrotImpl::GenerateImage(PPMImage& img) const {
    auto start = std::chrono::high_resolution_clock::now();
    for (long i = 0; i < img.x * img.y; i++) {
        double y0 = static_cast<double>(i) / static_cast<double>(img.x) / static_cast<double>(img.y) * (Y_MAX - Y_MIN) + Y_MIN;
        double x0 = static_cast<double>(i % img.x) / static_cast<double>(img.x) * (X_MAX - X_MIN) + X_MIN;
        img.data[i] = m_colors[GetIterations(x0, y0)];
    }
    auto end = std::chrono::high_resolution_clock::now();
    return std::chrono::duration_cast<std::chrono::microseconds>(end - start);
}

std::string CpuMandelbrotImpl::name() const {
    return "CPU";
}

CpuMandelbrotImpl::CpuMandelbrotImpl() {
    m_colors = new PPMPixel[MAX_ITERATION];
    for (int i = 0; i < MAX_ITERATION; i++) {
        m_colors[i] = GetColor((MAX_ITERATION - (double) i) / MAX_ITERATION);
    }
}

CpuMandelbrotImpl::~CpuMandelbrotImpl() {
    delete[] m_colors;
}
