#include "hip/hip_runtime.h"
#include <cmath>

#include "GpuMandelbrotImpl.cuh"
#include "../ColorImpl.cuh"

static constexpr size_t MAX_ITERATION = 32;
static constexpr double X_MIN = -2.01;
static constexpr double X_MAX = 0.48;
static constexpr double Y_MIN = -1.13;
static constexpr double Y_MAX = 1.13;

__constant__ PPMPixel d_colors[MAX_ITERATION]; // device array in constant memory

__device__ static unsigned int GetIterations(double x0, double y0) {
    double x = 0;
    double y = 0;
    int iteration = 0;
    while (x * x + y * y <= 4 && iteration < MAX_ITERATION) {
        double x_temp = x * x - y * y + x0;
        y = 2 * x * y + y0;
        x = x_temp;
        iteration++;
    }
    return iteration;
}

GpuMandelbrotImpl::GpuMandelbrotImpl() {
    auto* colors = new PPMPixel[MAX_ITERATION];
    for (int i = 0; i < MAX_ITERATION; i++) {
        colors[i] = GetColor((MAX_ITERATION - (double) i) / MAX_ITERATION);
    }
    hipMemcpyToSymbol(HIP_SYMBOL(d_colors), colors, MAX_ITERATION * sizeof(PPMPixel));
    delete[] colors;
}

GpuMandelbrotImpl::~GpuMandelbrotImpl() = default;

__global__ void mandelbrot(PPMImage img) {
    size_t row = (blockIdx.y * blockDim.y) + threadIdx.y;
    size_t col = (blockIdx.x * blockDim.x) + threadIdx.x;
    if ((row < img.y) && (col < img.x)) {
        size_t idx = row * img.x + col;
        double y0 = (double) (idx / img.x) / img.y * (Y_MAX - Y_MIN) + Y_MIN;
        double x0 = (double) (idx % img.x) / img.x * (X_MAX - X_MIN) + X_MIN;
        img.data[idx] = d_colors[GetIterations(x0, y0)];
    }
}

std::chrono::microseconds GpuMandelbrotImpl::GenerateImage(PPMImage& img) const {
    // GPU implementation
    hipEvent_t start, stop; //declare a start and stop event
    hipEventCreate(&start); //create both events
    hipEventCreate(&stop);

    size_t size = img.x * img.y * sizeof(PPMPixel);

    // device image
    PPMImage d_img;
    d_img.x = img.x;
    d_img.y = img.y;
    hipMalloc(&d_img.data, size);

    dim3 dimBlock(32, 32);
    dim3 dimGrid(std::ceil((float) d_img.x / 32), std::ceil((float) d_img.y / 32), 1);

    hipEventRecord(start);
    mandelbrot<<<dimGrid, dimBlock>>>(d_img);
    hipEventRecord(stop);

    hipMemcpy(img.data, d_img.data, size, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);

    float milliseconds = 0; //declare a variable to store runtime
    hipEventElapsedTime(&milliseconds, start, stop); //get the elapsed time

    return std::chrono::microseconds(static_cast<size_t>(milliseconds * std::chrono::microseconds::period::den /
                                                         std::chrono::milliseconds::period::den));
}

std::string GpuMandelbrotImpl::name() const {
    return "GPU Implementation";
}
